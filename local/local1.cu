#include <hip/hip_runtime.h>

#define NUMS 64
#define num_size 8
#define NUM 49


__global__ void local_1(float *a)
{
	float tmp[NUMS];
	int i;
	for(i=0;i<NUMS;i++)
	{
		tmp[i]=a[i];
	}
	for(i=0;i<NUMS;i++)
	{
		a[i]+=tmp[i];
	}
}

__global__ void local_2(float *a,float *b,float *c)
{
	float tmp_a[num_size*num_size];
	float temp;
	int i,j,k;
	for (i=0;i<num_size*num_size;i++)
	{
		tmp_a[i]=a[i];
	}
	for (i=0;i<num_size;i++)
	{
		for (j=0;j<num_size;j++)
		{
			temp=0.0;
			for (k=0;k<num_size;k++)
			{
				temp+=tmp_a[i*num_size+k]*b[k*num_size+j];
			}
			c[i*num_size+j]=temp;
		}
	}
}

__global__ void local_3(float *a)
{
	float tmp[NUM];
	float minf=0.0,temp;
	int mind;
	int i,j;
	for(i=0;i<NUM;i++)
	{
		tmp[i]=a[i];
	}
	for(i=0;i<NUM;i++)
	{
		minf=tmp[i];
		mind=i;
		for (j=i;j<NUM;j++)
		{
			if (minf>tmp[j])
			{
				minf=tmp[j];
				mind=i;
			}			
		}
		if (mind!=i)
		{
			temp=tmp[i];
			tmp[i]=tmp[mind];
			tmp[mind]=temp;
		}
	}
	a[0]=tmp[NUM-1];
}


