#include <hip/hip_runtime.h>

//#define NUMS 64
//#define num_size 8
//#define NUM 49

#define local_1(NUMS) \
__global__ void local_1_##NUMS(float *a) \
{\
	float tmp[NUMS];\
	int i;\
	for(i=0;i<NUMS;i++)\
	{\
		tmp[i]=a[i];\
	}\
	for(i=0;i<NUMS;i++)\
	{\
		a[i]+=tmp[i];\
	}\
}
//local_1(29)	//sm_10
//local_1(30)

//local_1(73)	//sm_35
//local_1(74)

#define local_2(num_size) \
	__global__ void local_2_##num_size(float *a,float *b,float *c)\
{\
	float tmp_a[num_size*num_size];\
	float temp;\
	int i,j,k;\
	for (i=0;i<num_size*num_size;i++)\
	{\
		tmp_a[i]=a[i];\
	}\
	for (i=0;i<num_size;i++)\
	{\
		for (j=0;j<num_size;j++)\
		{\
			temp=0.0;\
			for (k=0;k<num_size;k++)\
			{\
				temp+=tmp_a[i*num_size+k]*b[k*num_size+j];\
			}\
			c[i*num_size+j]=temp;\
		}\
	}\
}
//local_2(2) //sm_10
//local_2(3)
//local_2(4)
//local_2(5)
//local_2(6) //sm_35
//local_2(7)
//local_2(8)
//local_2(9)

#define local_3(NUM) \
__global__ void local_3_##NUM(float *a)\
{\
	float tmp[NUM];\
	float minf=0.0,temp;\
	int mind;\
	int i,j;\
	for(i=0;i<NUM;i++)\
	{\
		tmp[i]=a[i];\
	}\
	for(i=0;i<NUM;i++)\
	{\
		minf=tmp[i];\
		mind=i;\
		for (j=i;j<NUM;j++)\
		{\
			if (minf>tmp[j])\
			{\
				minf=tmp[j];\
				mind=i;\
			}			\
		}\
		if (mind!=i)\
		{\
			temp=tmp[i];\
			tmp[i]=tmp[mind];\
			tmp[mind]=temp;\
		}\
	}\
	a[0]=tmp[NUM-1];\
}
				//sm_10
local_3(2)
local_3(4)
local_3(8)
local_3(16)
local_3(32)
local_3(64)
local_3(128)
