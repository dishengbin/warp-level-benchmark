#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#define DATATYPE int
#define ARRAYLEN 1024*1024*256
#define REP 128
#define TIMETESTEVENT
#include <hip/hip_runtime.h>
#include "repeat.h"

__global__ void test_global_latency(double *time,DATATYPE *out,int its,DATATYPE *array)
{
	int p=0;
	double time_tmp=0.0;
	unsigned int start_time=0, stop_time=0;

//	for (int i=0;i<its;i++)										
	{									
		__syncthreads();
		start_time = clock();	
		repeat128(p=array[p];)
		stop_time = clock();
		time_tmp+=(stop_time-start_time);
	}
	time_tmp=time_tmp/128.0;
	out[0] =p;
	time[0] = time_tmp;
}

texture <int,1,hipReadModeElementType> texref;
__global__ void test_texture_latency(double *time,DATATYPE *out,int its)
{
	int p=0;
	double time_tmp=0.0;
	unsigned int start_time=0, stop_time=0;

//	for (int i=0;i<its;i++)										
	{									
		__syncthreads();
		start_time = clock();	
		repeat128(p=tex1Dfetch(texref,p);)
		stop_time = clock();
		time_tmp+=(stop_time-start_time);
	}
	time_tmp=time_tmp/128.0;
	out[1] =p;
	time[1] = time_tmp;
}



void call_test_latency(DATATYPE *h_array,DATATYPE *d_array,int step,int its,double *h_time,double *d_time,DATATYPE *d_out,DATATYPE *h_out)
{
	printf("111 111\n");

	if (hipSuccess != hipMemcpy(d_array,h_array,sizeof(DATATYPE)*ARRAYLEN,hipMemcpyHostToDevice)){ printf("1\n"); return; }

	printf("111 222\n");


	test_global_latency		<<<1,1>>>(d_time,d_out,its,d_array);
	if (hipDeviceSynchronize() != hipSuccess){
		printf("3\n");
		return;
	}
	printf("111 333\n");


	hipMemcpy(h_time,d_time,sizeof(double),hipMemcpyDeviceToHost);
	printf("%d:\t%f\t\n",step,h_time[0]);
//	printf("111 444\n");

}

