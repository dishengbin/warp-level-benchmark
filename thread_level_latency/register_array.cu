#include "hip/hip_runtime.h"
/************************************************************************/
/* writen by fang minquan (fmq@hpc6.com)                                */
/************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#define DATATYPE int
#define ARRAYLEN 2048
#define REP 128
//#define PRINTNEED
#define TIMETESTEVENT
#include <hip/hip_runtime.h>
#include "repeat.h"
__global__ void test_registerarray_latency(double *time,DATATYPE *out,int its,DATATYPE *array)
{
	DATATYPE register_array[4];
	int i;
	for (i=0;i<4;i++)
	{
		register_array[i]=(i+1)%4;
	}
	int p=0;
	double time_tmp=0.0;
	unsigned int start_time=0, stop_time=0;

	for (int i=0;i<its;i++)										
	{									
		__syncthreads();
		start_time = clock();	
		repeat128(p=register_array[p];)
		stop_time = clock();
		time_tmp+=(stop_time-start_time);
	}
	time_tmp=time_tmp/128.0/its;
	out[0] =p;
	time[0] = time_tmp;
}